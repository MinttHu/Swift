#include <iostream>
#include <vector>
#include "sys/time.h"
//#include "absl/random/random.h"

#include "coo2csc.h"
#include "Swift_cpu.h"
#include "gpu_csr2csc.h"
#include "gpu_cusparse_spmm.h"
#include "Swift_gpu_row_major.h"
#include "Swift_gpu_col_major.h"
#include "Swift.h"

#include "balance.h"
#include "block_catgory.h"

#include "ColSort_cuda.h"
#include "formatTransform_cuda.h"
#define BN 32

#ifndef VERIFYCSC
#define VERIFYCSC 1
#endif


int main(int argc, char ** argv)
{
    if(argc <2)
    {
        printf("error order\n");
        return 0;
    }

   int device_id = 0;
    // "Usage: ``./spmv -d 0 mtx A.mtx'' for Ax=y on device 0"
    int argi = 1;

    // load device id
    char *devstr;
    if(argc > argi)
    {
        devstr = argv[argi];
        argi++;
    }

    if (strcmp(devstr, "-d") != 0) return 0;

    if(argc > argi)
    {
        device_id = atoi(argv[argi]);
        argi++;
    }



    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);


    printf("---------------------------------------------------------------------------------------------\n");
    printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);


	char  *filename;
    filename = argv[3];


    int rowA,colA,nnz;
    int isSymmetricA;
    double *csrval;
    int *csrrowptr;
    int *csrcolidx;

    mmio_allinone(&rowA, &colA, &nnz, &isSymmetricA, &csrrowptr, &csrcolidx, &csrval ,filename);

//|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-
    std::string filePath(filename);

    size_t pos = filePath.find_last_of("/\\");

    std::string fileName;
    if (pos != std::string::npos) {
        fileName = filePath.substr(pos + 1);
    } else {
        fileName = filePath;
    }
    size_t dotPos = fileName.find_last_of('.');
    if (dotPos != std::string::npos) {
        fileName = fileName.substr(0, dotPos);
    }
    filename = new char[fileName.length() + 1];
    std::strcpy(filename, fileName.c_str());
//|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-|-

    printf("read success,input matrix A :(%i,%i) nnz =%i  width=%d \n",rowA,colA,nnz,BN);
  

    double *dense_matrix = (double *)malloc(sizeof(double) * colA*BN);
    memset(dense_matrix,0, sizeof(double) * colA*BN);

    unsigned seed;
    seed = time(0);
    srand(seed);
    for (int i=0; i<colA*BN; i++) 
    {
        dense_matrix[i] = double(rand() %100 - 50)/100;
    }

    for (int i=0; i<nnz; i++) 
    {
        csrval[i] = double(rand() %1000 - 500)/1000;
        //csrval[i] = i % 10 + 1;
    }


    double *golden_matrix_c = (double *)malloc(sizeof(double)*rowA *BN);
    memset(golden_matrix_c,0, sizeof(double) * rowA*BN);  



    for(int i=0; i<rowA;i++)
    {
      for(int j=csrrowptr[i]; j<csrrowptr[i+1]; j++)
      {
        for(int k=0; k<BN; k++)
        {
          int dense_index = csrcolidx[j] *  BN;
          golden_matrix_c[i*BN + k] += csrval[j] * dense_matrix[dense_index + k];
        }
      }
    }

    double *result_cusparse_spmm = (double *)malloc(sizeof(double)*rowA *BN);
    memset(result_cusparse_spmm,0, sizeof(double) * rowA*BN);
    double *result_cusparse_spmm1 = (double *)malloc(sizeof(double)*rowA *BN);
    memset(result_cusparse_spmm1,0, sizeof(double) * rowA*BN);
    double *dense_matrix1 = (double *)malloc(sizeof(double) * colA*BN);
    memset(dense_matrix1,0, sizeof(double) * colA*BN);

    dense_mtx2dense_mtx_spmm(colA, BN, dense_matrix, dense_matrix1);

    float time_cusparse_spmm = 0;
    float time_cusparse_spmm_pre=0;
    cusparse_spmm(time_cusparse_spmm_pre, time_cusparse_spmm, 
                  rowA, colA, nnz,
                  colA, BN, 
                  csrrowptr, csrcolidx, csrval,
                  dense_matrix1,
                  result_cusparse_spmm1);
    dense_mtx2dense_mtx_spmm(BN, rowA, result_cusparse_spmm1, result_cusparse_spmm);
#if VERIFYCSC
    int error_cusparse_spmm=0;
    ResultVerify(result_cusparse_spmm, golden_matrix_c, rowA*BN, error_cusparse_spmm);
    if(error_cusparse_spmm !=0)
    {
        printf("error cuSPARSE SpMM, error = %d\n", error_cusparse_spmm); 
        time_cusparse_spmm = -1111;
        time_cusparse_spmm_pre = -1111;
    }
    else
    {
        printf("success cuSPARSE SpMM,Time pre: %f ms, process:%f ms\n", time_cusparse_spmm_pre,time_cusparse_spmm);
    }
#endif



    int *cscrowidx = (int *)malloc(sizeof(int) * (nnz));
    memset(cscrowidx, 0, sizeof(int)*(nnz));


    int *csccolptr = (int *)malloc(sizeof(int) * (colA +1));
    memset(csccolptr, 0, sizeof(int)*(colA+1));

    int *nnzpercol=(int *)malloc(sizeof(int)*(colA));
    memset(nnzpercol, 0 ,sizeof(int)*(colA));


    double *cscval= (double *)malloc(sizeof(double)* (nnz));   
    memset(cscval, 0, sizeof(double)*(nnz));

    double csr2cscTime = 0;
    csr_to_csc(csr2cscTime, rowA, colA, nnz, csrval, csrrowptr, csrcolidx, cscval, cscrowidx, csccolptr, nnzpercol);

    double *testcsc_matrix_c = (double *)malloc(sizeof(double)*rowA *BN);
    memset(testcsc_matrix_c,0, sizeof(double) * rowA*BN); 

    for(int i=0; i<colA; i++)
    {
      for(int j=csccolptr[i]; j<csccolptr[i+1]; j++)
      {
        for(int k=0; k<BN; k++)
        {
          int row_index = cscrowidx[j];
          int dense_index = i * BN;
          testcsc_matrix_c[row_index *BN +k] += cscval[j] * dense_matrix[i *BN+k];
        }
      }
    }


#if VERIFYCSC
    int errorCSC=0;
    ResultVerify(testcsc_matrix_c, golden_matrix_c, rowA*BN, errorCSC);
    if(errorCSC !=0)
    {
        printf("error format csc, error = %d\n", errorCSC); 
    }
    else
    {
        printf("success format csc\n");
    }
#endif


    slide_matrix *matrixA = (slide_matrix *)malloc(sizeof(slide_matrix));

    int *sortrowidx_tmp = (int *)malloc(sizeof(int)*nnz);

    double *sortval_tmp = (double *)malloc(sizeof(double)*nnz);

    int *sortnnz_tmp= (int *)malloc(sizeof(int)*(colA+1));

    double *sort_dense_mtx = (double *)malloc(sizeof(double)*colA * BN);  
    memset(sort_dense_mtx,0,sizeof(double)*colA*BN);

double time_colsort = 0;

timeval tcolsort1, tcolsort2;

gettimeofday(&tcolsort1, NULL);

    col_sort(colA,
             BN,
             nnzpercol,
             csccolptr,
             cscrowidx,
             cscval,
              
             sortrowidx_tmp,
             sortval_tmp,
             sortnnz_tmp,
             
             dense_matrix,
             sort_dense_mtx);  


gettimeofday(&tcolsort2, NULL);  
time_colsort = (tcolsort2.tv_sec - tcolsort1.tv_sec) * 1000.0 + (tcolsort2.tv_usec - tcolsort1.tv_usec) / 1000.0;



double time_transform = 0;
double time_irrepart = 0;
timeval ttransform1, ttransform2;
gettimeofday(&ttransform1, NULL);
    
    double reside_ratio=0;
    formattransation(time_irrepart,
                     matrixA,
                     sortrowidx_tmp,
                     sortval_tmp,
                     sortnnz_tmp,
                     
                     nnz,
                     rowA,
                     colA,
                     reside_ratio);

    double shuffle_ratio = 0;
    block_catgory(matrixA,shuffle_ratio);

gettimeofday(&ttransform2, NULL); 
time_transform = (ttransform2.tv_sec - ttransform1.tv_sec) * 1000.0 + (ttransform2.tv_usec - ttransform1.tv_usec) / 1000.0;




    double *result_mtx = (double *)malloc(sizeof(double) * rowA*BN);
    memset(result_mtx,0, sizeof(double) * rowA*BN);
    double time_fastload_cpu=0;
    FastLoad_cpu(time_fastload_cpu,
                 matrixA,
                 nnz,
                 rowA,
                 colA,
                 BN,

                 sort_dense_mtx,
                 result_mtx);

#if VERIFYCSC
    int errorCPU=0;
    ResultVerify(result_mtx, golden_matrix_c, rowA*BN, errorCPU);
    if(errorCPU !=0)
    {
        printf("error FastLoad CPU, error = %d\n", errorCPU); 
    }
    else
    {
        printf("success FastLoad CPU time: %f ms\n",time_fastload_cpu);
    }
#endif




    float time_fastload_gpu=0;
    double gflops_fastload_gpu=0;

    float time_fastload_gpu1=0;
    double gflops_fastload_gpu1=0;

    float time_fastload_gpu2=0;
    double gflops_fastload_gpu2=0;
    memset(result_mtx,0, sizeof(double) * rowA*BN);     

    double *sort_dense_mtx1 = (double *)malloc(sizeof(double)*colA * BN);  
    memset(sort_dense_mtx1,0,sizeof(double)*colA*BN);
    dense_mtx2dense_mtx_spmm(colA, BN, sort_dense_mtx, sort_dense_mtx1);



    Swift_gpu1(filename,
               time_fastload_gpu,
               gflops_fastload_gpu,
               matrixA,
               rowA,
               colA,
               BN,
               nnz,
               sort_dense_mtx1,
               result_mtx,
               golden_matrix_c);

    memset(result_mtx,0, sizeof(double) * rowA*BN);

    int reside_n = matrixA->reside_col;
    int reside_nnz = matrixA->reside_nnz;
    int *reside_ptr = matrixA->reside_cscptr;
    int *reside_rowidx = matrixA->reside_cscrowidx;

double time_balance = 0;
timeval tbalance1, tbalance2;
gettimeofday(&tbalance1, NULL);

    balance(matrixA,
             rowA, reside_n, reside_nnz,
             reside_ptr,
             reside_rowidx);

gettimeofday(&tbalance2, NULL); 
time_balance = (tbalance2.tv_sec - tbalance1.tv_sec) * 1000.0 + (tbalance2.tv_usec - tbalance1.tv_usec) / 1000.0;



    Swift_gpu2(filename,
                  time_fastload_gpu1,
                  gflops_fastload_gpu1,
                  matrixA,
                  rowA,
                  colA,
                  BN,
                  nnz,
                  sort_dense_mtx,
                  result_mtx,
                  golden_matrix_c);

    memset(result_mtx,0, sizeof(double) * rowA*BN);


    Swift_GPU(filename,
              time_fastload_gpu2,
              gflops_fastload_gpu2,
              matrixA,
              rowA,
              colA,
              BN,
              nnz,
              sort_dense_mtx1,
              result_mtx,
              golden_matrix_c);



    float time_final;
    result_check(time_fastload_gpu, time_fastload_gpu1, time_fastload_gpu2, time_final);
  
    if (time_final != -1)
    {
     
        FILE *fout = fopen("data/results_spmm_32.txt", "a");
        if (fout == NULL)
            printf("Writing results fails.\n");
        fprintf(fout, "%s m %d n %d width %d nnz %d reside_ratio %f shuffle_ratio %f cuSPARSE: %f final %f \n",
            filename,rowA, colA, BN, nnz,reside_ratio,shuffle_ratio, time_cusparse_spmm,time_final);
        fclose(fout);
        
    }
    else
    {
        printf("FastLoad GPU SpMM Check NO PASS!\n");
        FILE *fout = fopen("data/results_spmm_32.txt", "a");
        if (fout == NULL)
            printf("Writing results fails.\n");
        fprintf(fout, "erro Swift (dense mtx col-major)%s \n",
                       filename );
        fclose(fout);
    }




    free(dense_matrix);
    free(dense_matrix1);
    free(golden_matrix_c);
    free(result_cusparse_spmm);
    free(result_cusparse_spmm1);
    free(cscrowidx);
    free(csccolptr);
    free(nnzpercol);
    free(cscval);
    free(testcsc_matrix_c);
    free(sortrowidx_tmp);
    free(sortval_tmp);
    free(sortnnz_tmp);
    //free(result_mtx);
    free(sort_dense_mtx); 
}